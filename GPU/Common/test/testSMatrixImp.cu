#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

/// \file testGPUSMatrixImp.cu
/// \author Matteo Concas, Maksym KIzitskyi

#define BOOST_TEST_MODULE Test GPUSMatrixImpl
#ifdef __HIPCC__
#define GPUPLATFORM "HIP"
#include "hip/hip_runtime.h"
#else
#define GPUPLATFORM "CUDA"
#include <hip/hip_runtime.h>
#endif

#include <iostream>
#include <boost/test/unit_test.hpp>
#include <MathUtils/SMatrixGPU.h>
#include <Math/SMatrix.h>
#include <random>

using MatSym3DGPU = o2::math_utils::SMatrixGPU<float, 3, 3, o2::math_utils::MatRepSymGPU<float, 3>>;
using MatSym3D = ROOT::Math::SMatrix<float, 3, 3, ROOT::Math::MatRepSym<float, 3>>;
using Mat3DGPU = o2::math_utils::SMatrixGPU<float, 3, 3, o2::math_utils::MatRepStdGPU<float, 3, 3>>;
using Mat3D = ROOT::Math::SMatrix<float, 3, 3, ROOT::Math::MatRepStd<float, 3, 3>>;

// Macro for checking CUDA errors
#define GPU_CHECK(call)                                                                      \
  do {                                                                                       \
    hipError_t error = call;                                                                \
    if (error != hipSuccess) {                                                              \
      fprintf(stderr, "CUDA Error: %s (error code %d)\n", hipGetErrorString(error), error); \
      return;                                                                                \
    }                                                                                        \
  } while (0)

namespace gpu
{
enum PrintMode {
  Decimal,
  Binary,
  Hexadecimal
};

__device__ void floatToBinaryString(float number, char* buffer)
{
  unsigned char* bytePointer = reinterpret_cast<unsigned char*>(&number);
  for (int byteIndex = 3; byteIndex >= 0; --byteIndex) {
    unsigned char byte = bytePointer[byteIndex];
    for (int bitIndex = 7; bitIndex >= 0; --bitIndex) {
      buffer[(3 - byteIndex) * 8 + (7 - bitIndex)] = (byte & (1 << bitIndex)) ? '1' : '0';
    }
  }
  buffer[32] = '\0'; // Null terminator
}

template <typename MatrixType>
GPUd() void printMatrix(const MatrixType& matrix, const char* name, const PrintMode mode)
{
  if (mode == PrintMode::Binary) {
    char buffer[33];
    for (int i = 0; i < 3; ++i) {
      for (int j = 0; j < 3; ++j) {
        floatToBinaryString(matrix(i, j), buffer);
        printf("%s(%d,%d) = %s\n", name, i, j, buffer);
      }
    }
  }
  if (mode == PrintMode::Decimal) {
    for (int i = 0; i < 3; ++i) {
      for (int j = 0; j < 3; ++j) {
        printf("%s(%i,%i) = %f\n", name, i, j, matrix(i, j));
      }
    }
  }
  if (mode == PrintMode::Hexadecimal) {
    for (int i = 0; i < 3; ++i) {
      for (int j = 0; j < 3; ++j) {
        printf("%s(%d,%d) = %x\n", name, i, j, o2::gpu::CAMath::Float2UIntReint(matrix(i, j)));
      }
    }
  }
}

// Invert test for symmetric matrix
template <typename T, int D>
__global__ void invertMatrixKernel(T* matrix)
{
  matrix->Invert();
}
} // namespace gpu

// Function to compare two matrices element-wise with a specified tolerance
template <typename MatrixType>
void compareMatricesElementWise(const MatrixType& mat1, const MatrixType& mat2, float tolerance)
{
  auto tol = boost::test_tools::tolerance(tolerance);

  for (unsigned int i = 0; i < mat1.kRows; ++i) {
    for (unsigned int j = 0; j < mat1.kCols; ++j) {
      BOOST_TEST(mat1(i, j) == mat2(i, j), tol);
    }
  }
}

// RAII class for CUDA resources
class GPUMemory
{
 public:
  GPUMemory(size_t size)
  {
    GPU_CHECK(hipMalloc(&device_ptr, size));
  }
  ~GPUMemory()
  {
    GPU_CHECK(hipFree(device_ptr));
  }
  void* get() const { return device_ptr; }

 private:
  void* device_ptr;
};

class GPUBenchmark
{
 public:
  GPUBenchmark()
  {
    GPU_CHECK(hipEventCreate(&startEvent));
    GPU_CHECK(hipEventCreate(&stopEvent));
  }

  ~GPUBenchmark()
  {
    GPU_CHECK(hipEventDestroy(startEvent));
    GPU_CHECK(hipEventDestroy(stopEvent));
  }

  void start()
  {
    GPU_CHECK(hipEventRecord(startEvent));
  }

  void stop()
  {
    GPU_CHECK(hipEventRecord(stopEvent));
    GPU_CHECK(hipEventSynchronize(stopEvent));
    GPU_CHECK(hipEventElapsedTime(&duration, startEvent, stopEvent));
  }

  float getDuration() const { return duration; }
  void printDuration() const
  {
    std::cout << "Kernel execution time: " << duration << " ms" << std::endl;
  }

 private:
  hipEvent_t startEvent, stopEvent;
  float duration;
};

template <typename T>
void discardResult(const T&)
{
}

void prologue()
{
  int deviceCount;
  hipError_t error = hipGetDeviceCount(&deviceCount);
  if (error != hipSuccess || !deviceCount) {
    std::cerr << "No " << GPUPLATFORM << " devices found" << std::endl;
    return;
  }

  for (int iDevice = 0; iDevice < deviceCount; ++iDevice) {
    hipDeviceProp_t deviceProp;
    discardResult(hipGetDeviceProperties(&deviceProp, iDevice));
    printf("%s Device %d: %s\n", GPUPLATFORM, iDevice, deviceProp.name);
  }
}

struct GPUSMatrixImplFixtureSolo {
  GPUSMatrixImplFixtureSolo() : SMatrixSym_d(sizeof(MatSym3DGPU)), SMatrixSym_h(), SMatrix_d(sizeof(Mat3DGPU)), SMatrix_h()
  {
    prologue();
    initializeMatrices();
    printMatrixSizes();
  }

  ~GPUSMatrixImplFixtureSolo() = default;
  void initializeMatrices()
  {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(1.0, 10.0);

    // Initialize host matrices with random values
    for (int i = 0; i < 3; ++i) {
      for (int j = i; j < 3; ++j) {
        SMatrixSym_h(i, j) = dis(gen);
        SMatrix_h(i, j) = dis(gen);
      }
    }
    SMatrixSym_original_h = SMatrixSym_h;
    SMatrix_original_h = SMatrix_h;

    // Copy host matrices to device
    GPU_CHECK(hipMemcpy(SMatrixSym_d.get(), &SMatrixSym_h, sizeof(MatSym3DGPU), hipMemcpyHostToDevice));
    GPU_CHECK(hipMemcpy(SMatrix_d.get(), &SMatrix_h, sizeof(Mat3DGPU), hipMemcpyHostToDevice));
  }

  void printMatrixSizes() const
  {
    printf("sizeof(MatSym3DGPU) = %zu bytes\n", sizeof(MatSym3DGPU));
    printf("sizeof(MatSym3D) = %zu bytes\n", sizeof(MatSym3D));
    printf("sizeof(Mat3DGPU) = %zu bytes\n", sizeof(Mat3DGPU));
    printf("sizeof(Mat3D) = %zu bytes\n", sizeof(Mat3D));
  }

  GPUMemory SMatrixSym_d;
  MatSym3D SMatrixSym_h;
  MatSym3D SMatrixSym_original_h;
  GPUMemory SMatrix_d;
  Mat3D SMatrix_h;
  Mat3D SMatrix_original_h;
};

BOOST_FIXTURE_TEST_CASE(MatrixInversion, GPUSMatrixImplFixtureSolo)
{
  float tolerance = 0.00001f;

  GPUBenchmark benchmark;
  benchmark.start();
  gpu::invertMatrixKernel<MatSym3DGPU, 3><<<1, 1>>>(static_cast<MatSym3DGPU*>(SMatrixSym_d.get()));
  benchmark.stop();
  benchmark.printDuration();
  discardResult(hipDeviceSynchronize());
  GPU_CHECK(hipGetLastError());
  GPU_CHECK(hipMemcpy(&SMatrixSym_h, SMatrixSym_d.get(), sizeof(MatSym3DGPU), hipMemcpyDeviceToHost));

  MatSym3D identitySym;
  identitySym(0, 0) = 1;
  identitySym(1, 1) = 1;
  identitySym(2, 2) = 1;
  auto operationSym = SMatrixSym_h * SMatrixSym_original_h;
  MatSym3D resultSym;
  ROOT::Math::AssignSym::Evaluate(resultSym, operationSym);
  compareMatricesElementWise(resultSym, identitySym, tolerance);

  benchmark.start();
  gpu::invertMatrixKernel<Mat3DGPU, 3><<<1, 1>>>(static_cast<Mat3DGPU*>(SMatrix_d.get()));
  benchmark.stop();
  benchmark.printDuration();
  discardResult(hipDeviceSynchronize());
  GPU_CHECK(hipGetLastError());
  GPU_CHECK(hipMemcpy(&SMatrix_h, SMatrix_d.get(), sizeof(Mat3DGPU), hipMemcpyDeviceToHost));

  Mat3D identity;
  identity(0, 0) = 1;
  identity(1, 1) = 1;
  identity(2, 2) = 1;
  auto operation = SMatrix_h * SMatrix_original_h;
  Mat3D result;
  ROOT::Math::Assign<float, 3, 3, decltype(operation), ROOT::Math::MatRepStd<float, 3, 3>, ROOT::Math::MatRepStd<float, 3, 3>>::Evaluate(result, operation);
  compareMatricesElementWise(result, identity, tolerance);
}

// struct GPUSMatrixImplFixtureDuo {
//   GPUSMatrixImplFixtureDuo() : i(3), SMatrixSym_d_A(sizeof(MatSym3DGPU)), SMatrixSym_h_A(), SMatrix_d_A(sizeof(Mat3DGPU)), SMatrix_h_A(), SMatrixSym_d_B(sizeof(MatSym3DGPU)), SMatrixSym_h_B(), SMatrix_d_B(sizeof(Mat3DGPU)), SMatrix_h_B()
//   {
//     prologue();
//     initializeMatrices();
//     printMatrixSizes();
//   }

//   ~GPUSMatrixImplFixtureDuo() = default;

//   void initializeMatrices()
//   {
//     std::random_device rd;
//     std::mt19937 gen(rd());
//     std::uniform_real_distribution<float> dis(1.0, 10.0);

//     // Initialize host matrices with random values
//     for (int i = 0; i < 3; ++i) {
//       for (int j = i; j < 3; ++j) {
//         SMatrixSym_h_A(i, j) = dis(gen);
//         SMatrix_h_A(i, j) = dis(gen);

//         SMatrixSym_h_B(i, j) = dis(gen);
//         SMatrix_h_B(i, j) = dis(gen);
//       }
//     }

//     // Copy host matrices to device
//     GPU_CHECK(hipMemcpy(SMatrixSym_d_A.get(), &SMatrixSym_h_A, sizeof(MatSym3DGPU), hipMemcpyHostToDevice));
//     GPU_CHECK(hipMemcpy(SMatrix_d_A.get(), &SMatrix_h_A, sizeof(Mat3DGPU), hipMemcpyHostToDevice));

//     GPU_CHECK(hipMemcpy(SMatrixSym_d_B.get(), &SMatrixSym_h_B, sizeof(MatSym3DGPU), hipMemcpyHostToDevice));
//     GPU_CHECK(hipMemcpy(SMatrix_d_B.get(), &SMatrix_h_B, sizeof(Mat3DGPU), hipMemcpyHostToDevice));
//   }

//   void printMatrixSizes() const
//   {
//     printf("sizeof(MatSym3DGPU) = %zu\n", sizeof(MatSym3DGPU));
//     printf("sizeof(MatSym3D) = %zu\n", sizeof(MatSym3D));
//     printf("sizeof(Mat3DGPU) = %zu\n", sizeof(Mat3DGPU));
//     printf("sizeof(Mat3D) = %zu\n", sizeof(Mat3D));
//   }

//   int i;
//   GPUMemory SMatrixSym_d_A;
//   MatSym3D SMatrixSym_h_A;

//   GPUMemory SMatrixSym_d_B;
//   MatSym3D SMatrixSym_h_B;

//   GPUMemory SMatrix_d_A;
//   Mat3D SMatrix_h_A;

//   GPUMemory SMatrix_d_B;
//   Mat3D SMatrix_h_B;
// };

// // Copy test for symmetric matrix
// template <typename T>
// __global__ void copySymMatrixKernel(
//   MatSym3DGPU* srcMatrix,
//   MatSym3DGPU* dstMatrix,
//   const PrintMode mode = PrintMode::Decimal)
// {
//   printf("\nStart copying general matrix\n");
//   printMatrix(*dstMatrix, "Before copying: ", mode);
//   printf("\nCopied values:\n");
//   printMatrix(*srcMatrix, "Copied values: ", mode);
//   printf("\nResult:\n");
//   *dstMatrix = *srcMatrix;
//   printMatrix(*dstMatrix, "After copying: ", mode);
//   printf("\n-------------------------------------------------------\n");
// }

// // Copy test for general matrix
// template <typename T>
// __global__ void copyMatrixKernel(
//   Mat3DGPU* srcMatrix,
//   Mat3DGPU* dstMatrix,
//   const PrintMode mode = PrintMode::Decimal)
// {
//   printf("\nStart copying general matrix\n");
//   printMatrix(*dstMatrix, "Before copying: ", mode);
//   printf("\nCopied values:\n");
//   printMatrix(*srcMatrix, "Copied values: ", mode);
//   printf("\nResult:\n");
//   *dstMatrix = *srcMatrix;
//   printMatrix(*dstMatrix, "After copying: ", mode);
//   printf("\n-------------------------------------------------------\n");
// }

// BOOST_FIXTURE_TEST_CASE(TestMatrixCopyingAndComparison, GPUSMatrixImplFixtureDuo)
// {
//   copySymMatrixKernel<float><<<1, 1>>>(static_cast<MatSym3DGPU*>(SMatrixSym_d_A.get()), static_cast<MatSym3DGPU*>(SMatrixSym_d_B.get()));
//   discardResult(hipDeviceSynchronize());
//   GPU_CHECK(hipGetLastError());

//   GPU_CHECK(hipMemcpy(&SMatrixSym_h_B, SMatrixSym_d_B.get(), sizeof(MatSym3DGPU), hipMemcpyDeviceToHost));

//   compareMatricesElementWise(SMatrixSym_h_A, SMatrixSym_h_B, 0.0);

//   copyMatrixKernel<float><<<1, 1>>>(static_cast<Mat3DGPU*>(SMatrix_d_A.get()), static_cast<Mat3DGPU*>(SMatrix_d_B.get()));
//   discardResult(hipDeviceSynchronize());
//   GPU_CHECK(hipGetLastError());

//   GPU_CHECK(hipMemcpy(&SMatrix_h_B, SMatrix_d_B.get(), sizeof(Mat3DGPU), hipMemcpyDeviceToHost));

//   compareMatricesElementWise(SMatrix_h_A, SMatrix_h_B, 0.0);
// }