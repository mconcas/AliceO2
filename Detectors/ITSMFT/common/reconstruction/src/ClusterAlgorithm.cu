#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

/// \file ClusterAlgorithm.cu
/// \brief Implementation of the Playne CCL Algorithm for Clustering on the GPU
/// \author Nikolaus Draeger [https://cds.cern.ch/record/2879828]

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <chrono>

#include "ITSMFTReconstruction/ClusterAlgorithm.h"
#include "ITSMFTReconstruction/BoundingBox.h"

using namespace o2::itsmft;

#define CHECK_CUDA_ERROR(err)                                       \
  do {                                                              \
    if (err != hipSuccess) {                                       \
      fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err)); \
    }                                                               \
  } while (0)

namespace o2::itsmft::gpu
{
using Point = o2::itsmft::Point;
using MinimalistBoundingBox = o2::itsmft::MinimalistBoundingBox;
using BoundingBox = o2::itsmft::BoundingBox;

__device__ void print(int* data, int* labels, int* parent, int* rank, int nrow, int ncol)
{
  for (int i = 0; i < nrow; i++) {
    for (int j = 0; j < ncol; j++) {
      // Print elements of the Data Matrix
      printf("%d ", data[i * ncol + j]);
    }
    printf("\t\t");

    for (int j = 0; j < ncol; j++) {
      // Print elements of the Labels Matrix
      printf("%d ", labels[i * ncol + j]);
    }
    printf("\t\t");

    for (int j = 0; j < ncol; j++) {
      // Print elements of the Parent Matrix
      printf("%d ", parent[i * ncol + j]);
    }
    printf("\t\t");

    for (int j = 0; j < ncol; j++) {
      // Print elements of the Rank Matrix
      printf("%d ", rank[i * ncol + j]);
    }
    printf("\n");
  }
  printf("\n");
}

// recursively find root of cluster
__device__ int find(int x, int* parent)
{
  if (x != parent[x]) {
    parent[x] = find(parent[x], parent);
  }
  return parent[x];
}

// merge two clusters
__device__ int unify(int x, int y, int* parent, int* rank)
{
  int rootX = find(x, parent);
  int rootY = find(y, parent);

  if (rootX == rootY)
    return;

  if (rank[rootX] < rank[rootY]) {
    parent[rootX] = rootY;
    return rootY;
  } else {
    parent[rootY] = rootX;
    if (rank[rootX] == rank[rootY]) {
      rank[rootX]++;
    }
    return rootX;
  }
}

__global__ void ccl_kernel(int N, int* data, int* coordinates, int* regionSizes, int* regionHeights, int* regionWidths,
                           int* startIndices, int* labels, int* parent, int* rank, int* numClusters,
                           int* clusterSizes, Point* scratchMemory, int* scratchMemIndex, int* stridedSizes, 
                           int* stridedPositions, int stride, MinimalistBoundingBox* boxes, MinimalistBoundingBox* stridedBoxes, 
                           int* chipIds, int* stridedChipIds)
{
  int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadIndex >= N) {
    return;
  }

  for (int regionIdx = threadIndex; regionIdx < N; regionIdx += blockDim.x * gridDim.x) {
    int startIndex = startIndices[regionIdx];
    int width = regionWidths[regionIdx];
    int height = regionHeights[regionIdx];
    int regionStartRow = coordinates[2 * regionIdx];
    int regionStartCol = coordinates[2 * regionIdx + 1];

    int currentLabel = 1;

    // First pass
    for (int r = 0; r < height; ++r) {
      for (int c = 0; c < width; ++c) {
        int idx = startIndex + r * width + c;
        if (data[idx] == 1) {
          int leftIdx = (c == 0 ? -1 : idx - 1);
          int topIdx = (r == 0 ? -1 : idx - width);

          bool connectLeft = leftIdx != -1 && data[leftIdx] == 1;
          bool connectTop = topIdx != -1 && data[topIdx] == 1;

          if (!connectLeft && !connectTop) {
            labels[idx] = currentLabel++;
            parent[idx] = idx;
            rank[idx] = 0;
            numClusters[regionIdx]++;
            clusterSizes[idx] = 1;
            boxes[idx].min_r = r + regionStartRow;
            boxes[idx].min_c = c + regionStartCol;
            boxes[idx].max_r = r + regionStartRow;
            boxes[idx].max_c = c + regionStartCol;
          } else if (connectLeft && connectTop) {
            int topRootIdx = find(topIdx, parent);
            int leftRootIdx = find(leftIdx, parent);

            if (topRootIdx == leftRootIdx) {
              labels[idx] = labels[topRootIdx];
              parent[idx] = topRootIdx;
              rank[idx] = 0;

              // in this case, the bounding box already encompasses the newly added pixel
            } else {
              int newRoot = unify(leftRootIdx, topRootIdx, parent, rank);
              labels[idx] = labels[newRoot];
              parent[idx] = newRoot;
              numClusters[regionIdx]--;
              clusterSizes[newRoot] = clusterSizes[leftRootIdx] + clusterSizes[topRootIdx];

              boxes[newRoot].min_r = min(boxes[topRootIdx].min_r, boxes[leftRootIdx].min_r);
              boxes[newRoot].min_c = min(boxes[topRootIdx].min_c, boxes[leftRootIdx].min_c);
              boxes[newRoot].max_r = max(boxes[topRootIdx].max_r, boxes[leftRootIdx].max_r);
              boxes[newRoot].max_c = max(boxes[topRootIdx].max_c, boxes[leftRootIdx].max_c);
            }
          } else {
            int rootIdx;
            if (connectLeft) {
              rootIdx = find(leftIdx, parent);
            } else if (connectTop) {
              rootIdx = find(topIdx, parent);
            }
            labels[idx] = labels[rootIdx];
            parent[idx] = rootIdx;
            rank[idx] = 0;
            clusterSizes[rootIdx]++;

            boxes[rootIdx].min_r = min(boxes[rootIdx].min_r, r + regionStartRow);
            boxes[rootIdx].min_c = min(boxes[rootIdx].min_c, c + regionStartCol);
            boxes[rootIdx].max_r = max(boxes[rootIdx].max_r, r + regionStartRow);
            boxes[rootIdx].max_c = max(boxes[rootIdx].max_c, c + regionStartCol);
          }
        }
      }
    }

    int localClusterIndex = 0;

    // Second pass
    for (int r = 0; r < height; ++r) {
      for (int c = 0; c < width; ++c) {
        int idx = startIndex + r * width + c;
        if (data[idx] != 0) {
          int rootIdx = find(idx, parent);
          labels[idx] = labels[rootIdx];

          if (idx == rootIdx) {
            int currentClusterSize = clusterSizes[idx];
            MinimalistBoundingBox currentClusterBox = boxes[idx];

            // global scratch memory index
            int reservedIndex = atomicAdd(scratchMemIndex, currentClusterSize);

            // pseudo local strided index
            int stridedIndex = stride * threadIndex + localClusterIndex;
            stridedSizes[stridedIndex] = currentClusterSize;
            stridedBoxes[stridedIndex] = currentClusterBox;
            stridedPositions[stridedIndex] = reservedIndex;
            stridedChipIds[stridedIndex] = chipIds[regionIdx];

            int localPixelIndex = 0;
            for (int r2 = 0; r2 < height; ++r2) {
              for (int c2 = 0; c2 < width; ++c2) {
                int idx2 = startIndex + r2 * width + c2;
                if (data[idx2] != 0 && find(idx2, parent) == rootIdx) {
                  scratchMemory[reservedIndex + localPixelIndex].r = r2 + regionStartRow;
                  scratchMemory[reservedIndex + localPixelIndex].c = c2 + regionStartCol;
                  localPixelIndex++;
                }
              }
            }
          }
          localClusterIndex++;
        }
      }
    }
  }
}
} // namespace o2::itsmft::gpu

std::vector<int> flatten(const std::vector<std::vector<std::vector<int>>>& data)
{
  std::vector<int> flatData;
  for (const auto& region : data)
    for (const auto& row : region)
      for (const auto& pixel : row)
        flatData.push_back(pixel);  
  return flatData;
}

std::vector<int> flattenCoordinates(const std::vector<std::pair<int,int>>& coordinates)
{
  std::vector<int> flatCoordinates;
  for (const auto& p : coordinates)
  {
    flatCoordinates.push_back(p.first);  
    flatCoordinates.push_back(p.second); 
  } 
  return flatCoordinates;
}

void ClusterAlgorithm::clusterize(const std::vector<std::vector<std::vector<int>>>& data, const std::vector<int>& chipIds, const std::vector<std::pair<int,int>>& coordinates, 
                                  std::vector<BoundingBox>& clusterBBoxes, std::vector<std::vector<PixelData>>& clusterPixels)
{
  auto start_all = std::chrono::high_resolution_clock::now();
  auto start_pre = std::chrono::high_resolution_clock::now();

  std::vector<int> flatData = flatten(data);
  std::vector<int> flatCoordinates = flattenCoordinates(coordinates); 

  int totalNumPixels = flatData.size();
  int numRegions = data.size();
  int scratchMemLength = 100 * numRegions; // assuming that there will not be more than 100 pixels in clusters per region on average
  int stride = 8;

  std::vector<int> regionSizes;
  std::vector<int> regionHeights;
  std::vector<int> regionWidths;
  std::vector<int> startIndices;
  int* parent = new int[totalNumPixels]();

  Point* scratchMemory = new Point[scratchMemLength]();
  int* stridedSizes = new int[numRegions * stride]();
  int* stridedPositions = new int[numRegions * stride]();
  int scratchMemIndex = 0;
  MinimalistBoundingBox* boxes = new MinimalistBoundingBox[totalNumPixels]();
  MinimalistBoundingBox* stridedBoxes = new MinimalistBoundingBox[numRegions * stride]();
  int* stridedChipIds = new int[numRegions * stride]();

  auto start_dataprep = std::chrono::high_resolution_clock::now();

  for (int i = 0; i < totalNumPixels; i++) {
    parent[i] = i;
  }

  startIndices.push_back(0);

  for (const auto& region : data) {
    regionSizes.push_back(region.size() * (region.empty() ? 0 : region[0].size()));
    regionHeights.push_back(region.size());
    regionWidths.push_back((region.empty() ? 0 : region[0].size()));
    startIndices.push_back(startIndices.back() + regionSizes.back());
  }

  // Remove the last start index (beyond the end of the data)
  startIndices.pop_back();

  auto stop_dataprep = std::chrono::high_resolution_clock::now();

  int* deviceData;
  int* deviceChipIds;
  int* deviceSizes;
  int* deviceHeights;
  int* deviceWidths;
  int* deviceStartIndices;
  int* deviceLabels;
  int* deviceParents;
  int* deviceRanks;
  int* deviceClusterSizes;
  int* deviceNumClusters;
  int* deviceCoordinates;

  Point* deviceScratchMemory;
  int* deviceScratchMemIndex;
  int* deviceStridedSizes;
  int* deviceStridedPositions;
  int* deviceStridedChipIds;
  MinimalistBoundingBox* deviceBoxes;
  MinimalistBoundingBox* deviceStridedBoxes;

  auto start_malloc = std::chrono::high_resolution_clock::now();

  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceData, totalNumPixels * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceCoordinates, flatCoordinates.size() * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceChipIds, numRegions * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceSizes, regionSizes.size() * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceHeights, regionHeights.size() * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceWidths, regionWidths.size() * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceStartIndices, startIndices.size() * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceLabels, totalNumPixels * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceParents, totalNumPixels * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceRanks, totalNumPixels * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceClusterSizes, totalNumPixels * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceNumClusters, numRegions * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceScratchMemory, scratchMemLength * sizeof(Point)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceScratchMemIndex, sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceStridedSizes, stride * numRegions * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceStridedPositions, stride * numRegions * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceStridedChipIds, stride * numRegions * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceBoxes, totalNumPixels * sizeof(MinimalistBoundingBox)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceStridedBoxes, stride * numRegions * sizeof(MinimalistBoundingBox)));

  auto stop_malloc = std::chrono::high_resolution_clock::now();

  // Add corresponding amount of bytes for each of the cudaMallocs from above
  int totalMallocBytes = 0;
  totalMallocBytes += flatCoordinates.size() * sizeof(int);
  totalMallocBytes += totalNumPixels * sizeof(int);
  totalMallocBytes += numRegions * sizeof(int);
  totalMallocBytes += regionSizes.size() * sizeof(int);
  totalMallocBytes += regionHeights.size() * sizeof(int);
  totalMallocBytes += regionWidths.size() * sizeof(int);
  totalMallocBytes += startIndices.size() * sizeof(int);
  totalMallocBytes += totalNumPixels * sizeof(int);
  totalMallocBytes += totalNumPixels * sizeof(int);
  totalMallocBytes += totalNumPixels * sizeof(int);
  totalMallocBytes += totalNumPixels * sizeof(int);
  totalMallocBytes += numRegions * sizeof(int);
  totalMallocBytes += scratchMemLength * sizeof(Point);
  totalMallocBytes += sizeof(int);
  totalMallocBytes += stride * numRegions * sizeof(int);
  totalMallocBytes += stride * numRegions * sizeof(int);
  totalMallocBytes += stride * numRegions * sizeof(int);
  totalMallocBytes += totalNumPixels * sizeof(MinimalistBoundingBox);
  totalMallocBytes += stride * numRegions * sizeof(MinimalistBoundingBox);

  auto start_memcpy = std::chrono::high_resolution_clock::now();

  CHECK_CUDA_ERROR(hipMemcpy(deviceData, flatData.data(), totalNumPixels * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceCoordinates, flatCoordinates.data(), flatCoordinates.size() * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceChipIds, chipIds.data(), numRegions * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceSizes, regionSizes.data(), regionSizes.size() * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceWidths, regionWidths.data(), regionWidths.size() * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceHeights, regionHeights.data(), regionHeights.size() * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceStartIndices, startIndices.data(), startIndices.size() * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceParents, parent, totalNumPixels * sizeof(int), hipMemcpyHostToDevice));

  auto stop_memcpy = std::chrono::high_resolution_clock::now();

  // Add corresponding amount of bytes for each of the cudaMemcpys from above
  int totalMemcpyBytes = 0;
  totalMemcpyBytes += flatCoordinates.size() * sizeof(int);
  totalMemcpyBytes += totalNumPixels * sizeof(int);
  totalMemcpyBytes += numRegions * sizeof(int);
  totalMemcpyBytes += regionSizes.size() * sizeof(int);
  totalMemcpyBytes += regionHeights.size() * sizeof(int);
  totalMemcpyBytes += regionWidths.size() * sizeof(int);
  totalMemcpyBytes += startIndices.size() * sizeof(int);
  totalMemcpyBytes += totalNumPixels * sizeof(int);

  auto start_memset = std::chrono::high_resolution_clock::now();

  CHECK_CUDA_ERROR(hipMemset(deviceLabels, 0, totalNumPixels * sizeof(int)));
  CHECK_CUDA_ERROR(hipMemset(deviceRanks, 0, totalNumPixels * sizeof(int)));
  CHECK_CUDA_ERROR(hipMemset(deviceClusterSizes, 0, totalNumPixels * sizeof(int)));
  CHECK_CUDA_ERROR(hipMemset(deviceNumClusters, 0, numRegions * sizeof(int)));
  CHECK_CUDA_ERROR(hipMemset(deviceScratchMemory, 0, scratchMemLength * sizeof(Point)));
  CHECK_CUDA_ERROR(hipMemset(deviceScratchMemIndex, 0, sizeof(int)));
  CHECK_CUDA_ERROR(hipMemset(deviceStridedSizes, 0, stride * numRegions * sizeof(int)));
  CHECK_CUDA_ERROR(hipMemset(deviceStridedPositions, 0, stride * numRegions * sizeof(int)));
  CHECK_CUDA_ERROR(hipMemset(deviceStridedChipIds, 0, stride * numRegions * sizeof(int)));
  CHECK_CUDA_ERROR(hipMemset(deviceBoxes, 0, totalNumPixels * sizeof(MinimalistBoundingBox)));
  CHECK_CUDA_ERROR(hipMemset(deviceStridedBoxes, 0, stride * numRegions * sizeof(MinimalistBoundingBox)));

  auto stop_memset = std::chrono::high_resolution_clock::now();

  // Add corresponding amount of bytes for each of the cudaMemsets from above
  int totalMemsetBytes = 0;
  totalMemsetBytes += totalNumPixels * sizeof(int);
  totalMemsetBytes += totalNumPixels * sizeof(int);
  totalMemsetBytes += totalNumPixels * sizeof(int);
  totalMemsetBytes += numRegions * sizeof(int);
  totalMemsetBytes += scratchMemLength * sizeof(Point);
  totalMemsetBytes += sizeof(int);
  totalMemsetBytes += stride * numRegions * sizeof(int);
  totalMemsetBytes += stride * numRegions * sizeof(int);
  totalMemsetBytes += stride * numRegions * sizeof(int);
  totalMemsetBytes += totalNumPixels * sizeof(MinimalistBoundingBox);
  totalMemsetBytes += stride * numRegions * sizeof(MinimalistBoundingBox);

  const int threadsPerBlock = 256;
  const int numBlocks = (numRegions + threadsPerBlock - 1) / threadsPerBlock;

  auto end_pre = std::chrono::high_resolution_clock::now();

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  gpu::ccl_kernel<<<numBlocks, threadsPerBlock>>>(numRegions, deviceData, deviceCoordinates, deviceSizes, deviceHeights,
                                                  deviceWidths, deviceStartIndices, deviceLabels, deviceParents, deviceRanks,
                                                  deviceNumClusters, deviceClusterSizes, deviceScratchMemory, deviceScratchMemIndex,
                                                  deviceStridedSizes, deviceStridedPositions, stride, deviceBoxes, deviceStridedBoxes, 
                                                  deviceChipIds, deviceStridedChipIds);
  hipEventRecord(stop);

  auto start_post = std::chrono::high_resolution_clock::now();

  CHECK_CUDA_ERROR(hipMemcpy(scratchMemory, deviceScratchMemory, scratchMemLength * sizeof(int), hipMemcpyDeviceToHost)); // copying can be shortened for tiny optimization
  CHECK_CUDA_ERROR(hipMemcpy(stridedSizes, deviceStridedSizes, stride * numRegions * sizeof(int), hipMemcpyDeviceToHost));
  CHECK_CUDA_ERROR(hipMemcpy(stridedPositions, deviceStridedPositions, stride * numRegions * sizeof(int), hipMemcpyDeviceToHost));
  CHECK_CUDA_ERROR(hipMemcpy(stridedChipIds, deviceStridedChipIds, stride * numRegions * sizeof(int), hipMemcpyDeviceToHost));
  CHECK_CUDA_ERROR(hipMemcpy(stridedBoxes, deviceStridedBoxes, stride * numRegions * sizeof(MinimalistBoundingBox), hipMemcpyDeviceToHost));

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout << "GPU KERNEL TOOK " << milliseconds << "ms" << std::endl;

  for (int clusterIdx = 0; clusterIdx < stride * numRegions; ++clusterIdx) {
    if (stridedSizes[clusterIdx] == 0)
      continue;

    clusterPixels.emplace_back();
    int clusterStartPosition = stridedPositions[clusterIdx];
    int chipId = stridedChipIds[clusterIdx];

    const MinimalistBoundingBox& sBox = stridedBoxes[clusterIdx];
    BoundingBox bBox(chipId);
    bBox.rowMin = static_cast<uint16_t>(sBox.min_r);
    bBox.colMin = static_cast<uint16_t>(sBox.min_c);
    bBox.rowMax = static_cast<uint16_t>(sBox.max_r);
    bBox.colMax = static_cast<uint16_t>(sBox.max_c);
    clusterBBoxes.push_back(bBox);

    for (int pixelIdx = 0; pixelIdx < stridedSizes[clusterIdx]; ++pixelIdx) {
      clusterPixels.back().push_back(PixelData(scratchMemory[clusterStartPosition + pixelIdx].r, scratchMemory[clusterStartPosition + pixelIdx].c));
    }
  }

  delete[] parent;
  delete[] scratchMemory;
  delete[] stridedSizes;
  delete[] stridedPositions;
  delete[] stridedChipIds;
  delete[] boxes;
  delete[] stridedBoxes;

  CHECK_CUDA_ERROR(hipFree(deviceData));
  CHECK_CUDA_ERROR(hipFree(deviceCoordinates));
  CHECK_CUDA_ERROR(hipFree(deviceChipIds));
  CHECK_CUDA_ERROR(hipFree(deviceSizes));
  CHECK_CUDA_ERROR(hipFree(deviceHeights));
  CHECK_CUDA_ERROR(hipFree(deviceWidths));
  CHECK_CUDA_ERROR(hipFree(deviceStartIndices));
  CHECK_CUDA_ERROR(hipFree(deviceLabels));
  CHECK_CUDA_ERROR(hipFree(deviceParents));
  CHECK_CUDA_ERROR(hipFree(deviceRanks));
  CHECK_CUDA_ERROR(hipFree(deviceClusterSizes));
  CHECK_CUDA_ERROR(hipFree(deviceNumClusters));
  CHECK_CUDA_ERROR(hipFree(deviceScratchMemory));
  CHECK_CUDA_ERROR(hipFree(deviceScratchMemIndex));
  CHECK_CUDA_ERROR(hipFree(deviceStridedSizes));
  CHECK_CUDA_ERROR(hipFree(deviceStridedPositions));
  CHECK_CUDA_ERROR(hipFree(deviceStridedChipIds));
  CHECK_CUDA_ERROR(hipFree(deviceBoxes));
  CHECK_CUDA_ERROR(hipFree(deviceStridedBoxes));

  auto end_post = std::chrono::high_resolution_clock::now();
  auto end_all = std::chrono::high_resolution_clock::now();

  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_all - start_all);
  std::cout << "Time taken for all: " << duration.count() / 1000 << " ms" << std::endl;

  duration = std::chrono::duration_cast<std::chrono::microseconds>(end_pre - start_pre);
  std::cout << "Time taken for pre: " << duration.count() / 1000 << " ms" << std::endl;

  duration = std::chrono::duration_cast<std::chrono::microseconds>(end_post - start_post);
  std::cout << "Time taken for post: " << duration.count() / 1000 << " ms" << std::endl;

  duration = std::chrono::duration_cast<std::chrono::microseconds>(stop_dataprep - start_dataprep);
  std::cout << "Time taken for dataprep: " << duration.count() / 1000 << " ms" << std::endl;

  std::cout << std::endl;

  auto malloctime = std::chrono::duration_cast<std::chrono::microseconds>(stop_malloc - start_malloc);
  std::cout << "Time taken for MALLOC: " << malloctime.count() << " us" << std::endl;
  std::cout << "Bytes allocated using MALLOC: " << totalMallocBytes << std::endl;
  double throughput_malloc = static_cast<double>(totalMallocBytes) / (malloctime.count() / 1e6) / 1e9;
  std::cout << "Resulting Throughput (GB/s): " << throughput_malloc << std::endl;
  std::cout << std::endl;

  auto memcpytime = std::chrono::duration_cast<std::chrono::microseconds>(stop_memcpy - start_memcpy);
  std::cout << "Time taken for MEMCPY: " << memcpytime.count() << " us" << std::endl;
  std::cout << "Bytes set using MEMCPY: " << totalMemcpyBytes << std::endl;
  double throughput_memcpy = static_cast<double>(totalMemcpyBytes) / (memcpytime.count() / 1e6) / 1e9;
  std::cout << "Resulting Throughput (GB/s): " << throughput_memcpy << std::endl;
  std::cout << std::endl;

  auto memsettime = std::chrono::duration_cast<std::chrono::microseconds>(stop_memset - start_memset);
  std::cout << "Time taken for MEMSET: " << memsettime.count() << " us" << std::endl;
  std::cout << "Bytes set using MEMSET: " << totalMemsetBytes << std::endl;
  double throughput_memset = static_cast<double>(totalMemsetBytes) / (memsettime.count() / 1e6) / 1e9;
  std::cout << "Resulting Throughput (GB/s): " << throughput_memset << std::endl;
  std::cout << std::endl;
}