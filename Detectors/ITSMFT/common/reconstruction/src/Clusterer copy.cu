#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

/// \file Clusterer.cxx
/// \brief Implementation of the ITS cluster finder
#include <algorithm>
#include <TTree.h>
// #include "Framework/Logger.h"
// #include "ITSMFTBase/GeometryTGeo.h"
#include "ITSMFTReconstruction/Clusterer.h"
#include "SimulationDataFormat/MCTruthContainer.h"
#include "CommonDataFormat/InteractionRecord.h"
#include <hip/hip_runtime.h>

#ifdef WITH_OPENMP
#include <omp.h>
#endif
using namespace o2::itsmft;

namespace o2::itsmft::gpu
{
// Empty CUDA kernel
__global__ void empty_kernel()
{
  printf("Hello world from gpu\n");
}
} // namespace o2::itsmft::gpu

//__________________________________________________
void Clusterer::process(int nThreads, PixelReader& reader, CompClusCont* compClus,
                        PatternCont* patterns, ROFRecCont* vecROFRec, MCTruth* labelsCl)
{
#ifdef _PERFORM_TIMING_
  mTimer.Start(kFALSE);
#endif
  if (nThreads < 1) {
    nThreads = 1;
  }
  auto autoDecode = reader.getDecodeNextAuto();
  int rofcount{0};
  do {
    if (autoDecode) {
      reader.setDecodeNextAuto(false); // internally do not autodecode
      if (!reader.decodeNextTrigger()) {
        break;                         // on the fly decoding was requested, but there were no data left
      }
    }
    if (reader.getInteractionRecord().isDummy()) {
      continue; // No IR info was found
    }
    // pre-fetch all non-empty chips of current ROF
    ChipPixelData* curChipData = nullptr;
    mFiredChipsPtr.clear();

    // NOTE: should be total number of pixels fired in ROF
    size_t nPix = 0;

    // NOTE: cycle until no more chips fired in ROF 

    // NOTE: not really sure how getNextChipData works tbh. should not matter for now
    // NOTE: does it only return chips that fired?
    while ((curChipData = reader.getNextChipData(mChips))) {
      mFiredChipsPtr.push_back(curChipData);
      nPix += curChipData->getData().size();
    }

    auto& rof = vecROFRec->emplace_back(reader.getInteractionRecord(), vecROFRec->size(), compClus->size(), 0); // create new ROF

    uint16_t nFired = mFiredChipsPtr.size();
    if (!nFired) {
      if (autoDecode) {
        continue;
      }
      break; // just 1 ROF was asked to be processed
    }
    // NOTE: probably start (at least) one thread per chip. less threads needed if less chips fired
    if (nFired < nThreads) {
      nThreads = nFired;
    }
#ifndef WITH_OPENMP
    nThreads = 1;
#endif
    uint16_t chipStep = nThreads > 1 ? (nThreads == 2 ? 20 : 10) : nFired;
    int dynGrp = std::min(4, std::max(1, nThreads / 2));
    if (nThreads > mThreads.size()) {
      int oldSz = mThreads.size();
      mThreads.resize(nThreads);
      for (int i = oldSz; i < nThreads; i++) {
        mThreads[i] = std::make_unique<ClustererThread>(this, i);
      }
    }
#ifdef WITH_OPENMP
#pragma omp parallel for schedule(dynamic, dynGrp) num_threads(nThreads)
    //>> start of MT region
    for (uint16_t ic = 0; ic < nFired; ic += chipStep) {
      auto ith = omp_get_thread_num();
      if (nThreads > 1) {
        mThreads[ith]->process(ic, std::min(chipStep, uint16_t(nFired - ic)),
                               &mThreads[ith]->compClusters,
                               patterns ? &mThreads[ith]->patterns : nullptr,
                               labelsCl ? reader.getDigitsMCTruth() : nullptr,
                               labelsCl ? &mThreads[ith]->labels : nullptr, rof);
      } else { // put directly to the destination
        mThreads[0]->process(0, nFired, compClus, patterns, labelsCl ? reader.getDigitsMCTruth() : nullptr, labelsCl, rof);
      }
    }
    //<< end of MT region
#else
    mThreads[0]->process(0, nFired, compClus, patterns, labelsCl ? reader.getDigitsMCTruth() : nullptr, labelsCl, rof);
#endif
    // copy data of all threads but the 1st one to final destination
    if (nThreads > 1) {
#ifdef _PERFORM_TIMING_
      mTimerMerge.Start(false);
#endif
      size_t nClTot = 0, nPattTot = 0;
      int chid = 0, thrStatIdx[nThreads];
      for (int ith = 0; ith < nThreads; ith++) {
        std::sort(mThreads[ith]->stats.begin(), mThreads[ith]->stats.end(), [](const ThreadStat& a, const ThreadStat& b) { return a.firstChip < b.firstChip; });
        thrStatIdx[ith] = 0;
        nClTot += mThreads[ith]->compClusters.size();
        nPattTot += mThreads[ith]->patterns.size();
      }
      compClus->reserve(nClTot);
      if (patterns) {
        patterns->reserve(nPattTot);
      }
      while (chid < nFired) {
        for (int ith = 0; ith < nThreads; ith++) {
          if (thrStatIdx[ith] >= mThreads[ith]->stats.size()) {
            continue;
          }
          const auto& stat = mThreads[ith]->stats[thrStatIdx[ith]];
          if (stat.firstChip == chid) {
            thrStatIdx[ith]++;
            chid += stat.nChips; // next chip to look
            const auto clbeg = mThreads[ith]->compClusters.begin() + stat.firstClus;
            auto szold = compClus->size();
            compClus->insert(compClus->end(), clbeg, clbeg + stat.nClus);
            if (patterns) {
              const auto ptbeg = mThreads[ith]->patterns.begin() + stat.firstPatt;
              patterns->insert(patterns->end(), ptbeg, ptbeg + stat.nPatt);
            }
            if (labelsCl) {
              labelsCl->mergeAtBack(mThreads[ith]->labels, stat.firstClus, stat.nClus);
            }
          }
        }
      }
      for (int ith = 0; ith < nThreads; ith++) {
        mThreads[ith]->patterns.clear();
        mThreads[ith]->compClusters.clear();
        mThreads[ith]->labels.clear();
        mThreads[ith]->stats.clear();
      }
#ifdef _PERFORM_TIMING_
      mTimerMerge.Stop();
#endif
    } else {
      mThreads[0]->stats.clear();
    }
    rof.setNEntries(compClus->size() - rof.getFirstEntry()); // update
  } while (autoDecode);
  reader.setDecodeNextAuto(autoDecode);                      // restore setting
#ifdef _PERFORM_TIMING_
  mTimer.Stop();
  LOGP(info, "Time to finish: {}s", mTimer.RealTime());
#endif
  std::vector<Clusterer::BBox> outClusterBBoxes;
  std::vector<std::vector<PixelData>> outClusterPixels;
  mToyProblem.executeToyProblem(outClusterBBoxes, outClusterPixels);
  std::cout << "TOY PROBLEM PRODUCED: " << outClusterBBoxes.size() << " BOUNDING BOXES." << std::endl;
  std::cout << "TOY PROBLEM PRODUCED: " << outClusterPixels.size() << " PIXEL LISTS." << std::endl;
}

//__________________________________________________
void Clusterer::ClustererThread::process(uint16_t chip, uint16_t nChips, CompClusCont* compClusPtr, PatternCont* patternsPtr,
                                         const ConstMCTruth* labelsDigPtr, MCTruth* labelsClPtr, const ROFRecord& rofPtr)
{
  if (stats.empty() || stats.back().firstChip + stats.back().nChips != chip) { // there is a jump, register new block
    stats.emplace_back(ThreadStat{chip, 0, uint32_t(compClusPtr->size()), patternsPtr ? uint32_t(patternsPtr->size()) : 0, 0, 0});
  }
  for (int ic = 0; ic < nChips; ic++) {
    auto* curChipData = parent->mFiredChipsPtr[chip + ic];
    #pragma omp critical
    {
      parent->mToyProblem.addChipAsync(curChipData);
    }
    auto chipID = curChipData->getChipID();
    if (parent->mMaxBCSeparationToMask > 0) { // mask pixels fired from the previous ROF
      const auto& chipInPrevROF = parent->mChipsOld[chipID];
      if (std::abs(rofPtr.getBCData().differenceInBC(chipInPrevROF.getInteractionRecord())) < parent->mMaxBCSeparationToMask) {
        parent->mMaxRowColDiffToMask ? curChipData->maskFiredInSample(parent->mChipsOld[chipID], parent->mMaxRowColDiffToMask) : curChipData->maskFiredInSample(parent->mChipsOld[chipID]);
      }
    }
    auto nclus0 = compClusPtr->size();
    auto validPixID = curChipData->getFirstUnmasked();
    auto npix = curChipData->getData().size();
    if (validPixID < npix) {    // chip data may have all of its pixels masked!
      auto valp = validPixID++;
      if (validPixID == npix) { // special case of a single pixel fired on the chip
        finishChipSingleHitFast(valp, curChipData, compClusPtr, patternsPtr, labelsDigPtr, labelsClPtr);
      } else {
        initChip(curChipData, valp);
        for (; validPixID < npix; validPixID++) {
          if (!curChipData->getData()[validPixID].isMasked()) {
            updateChip(curChipData, validPixID);
          }
        }
        finishChip(curChipData, compClusPtr, patternsPtr, labelsDigPtr, labelsClPtr);
      }
    }
    if (parent->mMaxBCSeparationToMask > 0) { // current chip data will be used in the next ROF to mask overflow pixels
      parent->mChipsOld[chipID].swap(*curChipData);
    }
  }
  auto& currStat = stats.back();
  currStat.nChips += nChips;
  currStat.nClus = compClusPtr->size() - currStat.firstClus;
  currStat.nPatt = patternsPtr ? (patternsPtr->size() - currStat.firstPatt) : 0;
}

//__________________________________________________
void Clusterer::ClustererThread::finishChip(ChipPixelData* curChipData, CompClusCont* compClusPtr,
                                            PatternCont* patternsPtr, const ConstMCTruth* labelsDigPtr, MCTruth* labelsClusPtr)
{
  const auto& pixData = curChipData->getData();
  for (int i1 = 0; i1 < preClusterHeads.size(); ++i1) {
    auto ci = preClusterIndices[i1];
    if (ci < 0) {
      continue;
    }
    BBox bbox(curChipData->getChipID());
    int nlab = 0;
    int next = preClusterHeads[i1];
    pixArrBuff.clear();
    while (next >= 0) {
      const auto& pixEntry = pixels[next];
      const auto pix = pixData[pixEntry.second];
      pixArrBuff.push_back(pix); // needed for cluster topology
      bbox.adjust(pix.getRowDirect(), pix.getCol());
      if (labelsClusPtr) {
        if (parent->mSquashingDepth) { // the MCtruth for this pixel is stored in chip data: due to squashing we lose contiguity
          fetchMCLabels(curChipData->getOrderedPixId(pixEntry.second), labelsDigPtr, nlab);
        } else {                       // the MCtruth for this pixel is at curChipData->startID+pixEntry.second
          fetchMCLabels(pixEntry.second + curChipData->getStartID(), labelsDigPtr, nlab);
        }
      }
      next = pixEntry.first;
    }
    preClusterIndices[i1] = -1;
    for (int i2 = i1 + 1; i2 < preClusterHeads.size(); ++i2) {
      if (preClusterIndices[i2] != ci) {
        continue;
      }
      next = preClusterHeads[i2];
      while (next >= 0) {
        const auto& pixEntry = pixels[next];
        const auto pix = pixData[pixEntry.second]; // PixelData
        pixArrBuff.push_back(pix);                 // needed for cluster topology
        bbox.adjust(pix.getRowDirect(), pix.getCol());
        if (labelsClusPtr) {
          if (parent->mSquashingDepth) { // the MCtruth for this pixel is stored in chip data: due to squashing we lose contiguity
            fetchMCLabels(curChipData->getOrderedPixId(pixEntry.second), labelsDigPtr, nlab);
          } else {                       // the MCtruth for this pixel is at curChipData->startID+pixEntry.second
            fetchMCLabels(pixEntry.second + curChipData->getStartID(), labelsDigPtr, nlab);
          }
        }
        next = pixEntry.first;
      }
      preClusterIndices[i2] = -1;
    }
    if (bbox.isAcceptableSize()) {
      parent->streamCluster(pixArrBuff, &labelsBuff, bbox, parent->mPattIdConverter, compClusPtr, patternsPtr, labelsClusPtr, nlab);
    } else {
      auto warnLeft = MaxHugeClusWarn - parent->mNHugeClus;
      if (warnLeft > 0) {
        LOGP(warn, "Splitting a huge cluster: chipID {}, rows {}:{} cols {}:{}{}", bbox.chipID, bbox.rowMin, bbox.rowMax, bbox.colMin, bbox.colMax,
             warnLeft == 1 ? " (Further warnings will be muted)" : "");
#ifdef WITH_OPENMP
#pragma omp critical
#endif
        {
          parent->mNHugeClus++;
        }
      }
      BBox bboxT(bbox); // truncated box
      std::vector<PixelData> pixbuf;
      do {
        bboxT.rowMin = bbox.rowMin;
        bboxT.colMax = std::min(bbox.colMax, uint16_t(bboxT.colMin + o2::itsmft::ClusterPattern::MaxColSpan - 1));
        do { // Select a subset of pixels fitting the reduced bounding box
          bboxT.rowMax = std::min(bbox.rowMax, uint16_t(bboxT.rowMin + o2::itsmft::ClusterPattern::MaxRowSpan - 1));
          for (const auto& pix : pixArrBuff) {
            if (bboxT.isInside(pix.getRowDirect(), pix.getCol())) {
              pixbuf.push_back(pix);
            }
          }
          if (!pixbuf.empty()) { // Stream a piece of cluster only if the reduced bounding box is not empty
            parent->streamCluster(pixbuf, &labelsBuff, bboxT, parent->mPattIdConverter, compClusPtr, patternsPtr, labelsClusPtr, nlab, true);
            pixbuf.clear();
          }
          bboxT.rowMin = bboxT.rowMax + 1;
        } while (bboxT.rowMin < bbox.rowMax);
        bboxT.colMin = bboxT.colMax + 1;
      } while (bboxT.colMin < bbox.colMax);
    }
  }
}

//__________________________________________________
void Clusterer::ClustererThread::finishChipSingleHitFast(uint32_t hit, ChipPixelData* curChipData, CompClusCont* compClusPtr,
                                                         PatternCont* patternsPtr, const ConstMCTruth* labelsDigPtr, MCTruth* labelsClusPtr)
{
  auto pix = curChipData->getData()[hit];
  uint16_t row = pix.getRowDirect(), col = pix.getCol();

  if (labelsClusPtr) { // MC labels were requested
    int nlab = 0;
    fetchMCLabels(curChipData->getStartID() + hit, labelsDigPtr, nlab);
    auto cnt = compClusPtr->size();
    for (int i = nlab; i--;) {
      labelsClusPtr->addElement(cnt, labelsBuff[i]);
    }
  }

  // add to compact clusters, which must be always filled
  unsigned char patt[ClusterPattern::MaxPatternBytes]{0x1 << (7 - (0 % 8))}; // unrolled 1 hit version of full loop in finishChip
  uint16_t pattID = (parent->mPattIdConverter.size() == 0) ? CompCluster::InvalidPatternID : parent->mPattIdConverter.findGroupID(1, 1, patt);
  if ((pattID == CompCluster::InvalidPatternID || parent->mPattIdConverter.isGroup(pattID)) && patternsPtr) {
    patternsPtr->emplace_back(1); // rowspan
    patternsPtr->emplace_back(1); // colspan
    patternsPtr->insert(patternsPtr->end(), std::begin(patt), std::begin(patt) + 1);
  }
  compClusPtr->emplace_back(row, col, pattID, curChipData->getChipID());
}

//__________________________________________________
Clusterer::Clusterer() : mPattIdConverter()
{
  /*gpu::empty_kernel<<<1, 1>>>();

  // Check for any errors
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess)
    printf("Kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));*/
#ifdef _PERFORM_TIMING_
  mTimer.Stop();
  mTimer.Reset();
  mTimerMerge.Stop();
  mTimerMerge.Reset();
#endif
}

//__________________________________________________
void Clusterer::ClustererThread::initChip(const ChipPixelData* curChipData, uint32_t first)
{
  // init chip with the 1st unmasked pixel (entry "from" in the mChipData)
  prev = column1 + 1;
  curr = column2 + 1;
  resetColumn(curr);

  pixels.clear();
  preClusterHeads.clear();
  preClusterIndices.clear();
  auto pix = curChipData->getData()[first];
  currCol = pix.getCol();
  curr[pix.getRowDirect()] = 0; // can use getRowDirect since the pixel is not masked
  // start the first pre-cluster
  preClusterHeads.push_back(0);
  preClusterIndices.push_back(0);
  pixels.emplace_back(-1, first); // id of current pixel
  noLeftCol = true;               // flag that there is no column on the left to check yet
}

//__________________________________________________
void Clusterer::ClustererThread::updateChip(const ChipPixelData* curChipData, uint32_t ip)
{
  const auto pix = curChipData->getData()[ip];
  uint16_t row = pix.getRowDirect(); // can use getRowDirect since the pixel is not masked
  if (currCol != pix.getCol()) {     // switch the buffers
    swapColumnBuffers();
    resetColumn(curr);
    noLeftCol = false;
    if (pix.getCol() > currCol + 1) {
      // no connection with previous column, this pixel cannot belong to any of the
      // existing preclusters, create a new precluster and flag to check only the row above for next pixels of this column
      currCol = pix.getCol();
      addNewPrecluster(ip, row);
      noLeftCol = true;
      return;
    }
    currCol = pix.getCol();
  }

  Bool_t orphan = true;

  if (noLeftCol) {                              // check only the row above
    if (curr[row - 1] >= 0) {
      expandPreCluster(ip, row, curr[row - 1]); // attach to the precluster of the previous row
      return;
    }
  } else {
#ifdef _ALLOW_DIAGONAL_ALPIDE_CLUSTERS_
    int neighbours[]{curr[row - 1], prev[row], prev[row + 1], prev[row - 1]};
#else
    int neighbours[]{curr[row - 1], prev[row]};
#endif
    for (auto pci : neighbours) {
      if (pci < 0) {
        continue;
      }
      if (orphan) {
        expandPreCluster(ip, row, pci); // attach to the adjascent precluster
        orphan = false;
        continue;
      }
      // reassign precluster index to smallest one
      if (preClusterIndices[pci] < preClusterIndices[curr[row]]) {
        preClusterIndices[curr[row]] = preClusterIndices[pci];
      } else {
        preClusterIndices[pci] = preClusterIndices[curr[row]];
      }
    }
  }
  if (orphan) {
    addNewPrecluster(ip, row); // start new precluster
  }
}

//__________________________________________________
void Clusterer::ClustererThread::fetchMCLabels(int digID, const ConstMCTruth* labelsDig, int& nfilled)
{
  // transfer MC labels to cluster
  if (nfilled >= MaxLabels) {
    return;
  }
  const auto& lbls = labelsDig->getLabels(digID);
  for (int i = lbls.size(); i--;) {
    int ic = nfilled;
    for (; ic--;) { // check if the label is already present
      if (labelsBuff[ic] == lbls[i]) {
        return;     // label is found, do nothing
      }
    }
    labelsBuff[nfilled++] = lbls[i];
    if (nfilled >= MaxLabels) {
      break;
    }
  }
  //
}

//__________________________________________________
void Clusterer::clear()
{
  // reset
#ifdef _PERFORM_TIMING_
  mTimer.Stop();
  mTimer.Reset();
  mTimerMerge.Stop();
  mTimerMerge.Reset();
#endif
}

//__________________________________________________
void Clusterer::print() const
{
  // print settings
  LOGP(info, "Clusterizer squashes overflow pixels separated by {} BC and <= {} in row/col seeking down to {} neighbour ROFs", mMaxBCSeparationToSquash, mMaxRowColDiffToMask, mSquashingDepth);
  LOG(info) << "Clusterizer masks overflow pixels separated by < " << mMaxBCSeparationToMask << " BC and <= "
            << mMaxRowColDiffToMask << " in row/col";

#ifdef _PERFORM_TIMING_
  auto& tmr = const_cast<TStopwatch&>(mTimer); // ugly but this is what root does internally
  auto& tmrm = const_cast<TStopwatch&>(mTimerMerge);
  LOG(info) << "Inclusive clusterization timing (w/o disk IO): Cpu: " << tmr.CpuTime()
            << " Real: " << tmr.RealTime() << " s in " << tmr.Counter() << " slots";
  LOG(info) << "Threads output merging timing                : Cpu: " << tmrm.CpuTime()
            << " Real: " << tmrm.RealTime() << " s in " << tmrm.Counter() << " slots";

#endif
}

//__________________________________________________
void Clusterer::reset()
{
  // reset for new run
  clear();
  mNHugeClus = 0;
}



ToyProblem::ToyProblem(std::unique_ptr<RegionExtractor> regionExtractor, std::unique_ptr<ClusterAlgorithm> clusterAlgorithm)
  : regionExtractor(std::move(regionExtractor)), clusterAlgorithm(std::move(clusterAlgorithm))
{
}

void ToyProblem::addChip(ChipPixelData* chipData)
{
  if (!regionExtractor) {
    throw std::runtime_error("Clustering algorithm is not set");
  }
  std::vector<std::vector<std::vector<int>>> chipRegions = regionExtractor->preprocess(chipData, MAX_DIST_X, MAX_DIST_Y);
  // std::cout << "Added chip " << extractedRegions.size() + 1 << " of " << extractionTasks.size() << std::endl;
  extractedRegions.insert(extractedRegions.end(), chipRegions.begin(), chipRegions.end());
  chipIds.push_back(chipData->getChipID());
}

void ToyProblem::addChipAsync(ChipPixelData* chipData)
{
  extractionTasks.push_back([this, chipData] { addChip(chipData); });
}

void ToyProblem::executeExtractionAsync()
{
  if (extractionTasks.empty())
    return;
  std::cout << "Number of tasks: " << extractionTasks.size() << std::endl;
  Timer timer("executeExtractionAsync");
  for (auto& task : extractionTasks) {
    task();
  }
}

void ToyProblem::executeToyProblem(std::vector<Clusterer::BBox>& clusterBBoxes, std::vector<std::vector<PixelData>>& clusterPixels)
{
  Timer timer("executeToyProblem");
  std::cout << "starting extraction" << std::endl;
  executeExtractionAsync();
  std::cout << "extraction done, starting clustering" << std::endl;
  performClustering(clusterBBoxes, clusterPixels);
  std::cout << "clustering done, starting postprocessing" << std::endl;
  postProcess();
}

void ToyProblem::performClustering(std::vector<Clusterer::BBox>& clusterBBoxes, std::vector<std::vector<PixelData>>& clusterPixels)
{
  if (!clusterAlgorithm) {
    throw std::runtime_error("Clustering stragegy is not set");
  }
  Timer timer("performClustering");
  clusterAlgorithm->clusterize(extractedRegions, chipIds, clusterBBoxes, clusterPixels);
}

void ToyProblem::postProcess()
{
  Timer timer("postProcess");
}


std::vector<std::vector<std::vector<int>>> ExpansionRegionExtractor::preprocess(const ChipPixelData* chipData, const int maxdist_x, const int maxdist_y)
{
  std::vector<std::vector<std::vector<int>>> extractedRegions;
  std::vector<PixelData> pixelData = chipData->getData();

  // could be replaced by unordered_set for better runtime complexity
  // requires hash function for PixelData, however
  std::set<PixelData> pixelSet(pixelData.begin(), pixelData.end());

  std::vector<std::vector<int>> fullRegion = convertSparsePixelsToGrid(pixelData);
  
  int numRows = fullRegion.size();
  int numCols = 0;
  if (!fullRegion.empty()) {
    numCols = fullRegion[0].size();
  }

  std::vector<std::vector<PixelData*>> pixelDataPointers(numRows, std::vector<PixelData*>(numCols, nullptr));
  for (const PixelData& pixel : pixelSet) {
    const PixelData* ptr = &pixel;
    pixelDataPointers[pixel.getRow()][pixel.getCol()] = const_cast<PixelData*>(ptr);
  }

  while (!pixelSet.empty()) {
    PixelData nextPixel = *pixelSet.begin();
    int nextRow = nextPixel.getRow();
    int nextCol = nextPixel.getCol();
    Region regionInfo = {nextRow, nextCol, 1, 1};

    while (expandRegion(fullRegion, regionInfo, maxdist_x, maxdist_y, pixelSet, pixelDataPointers)) { }

    std::vector<std::vector<int>> region(regionInfo.height, std::vector<int>(regionInfo.width, 0));

    for (int i = 0; i < regionInfo.height; ++i) {
      for (int j = 0; j < regionInfo.width; ++j) {
        region[i][j] = fullRegion[i + regionInfo.row][j + regionInfo.col];
      }
    }

    for (int row = regionInfo.row; row < regionInfo.row + regionInfo.height; ++row) {
      for (int col = regionInfo.col; col < regionInfo.col + regionInfo.width; ++col) {

        if (row >= fullRegion.size() || col >= fullRegion[row].size()) {
          std::cout << "ERROR: Trying to access out of bounds index in fullRegion" << std::endl;
        }

        if (fullRegion[row][col] != 1) continue;

        fullRegion[row][col] = 0;

        if (row >= pixelDataPointers.size() || col >= pixelDataPointers[row].size()) {
          std::cout << "ERROR: Trying to access out of bounds index in pixelDataPointers" << std::endl;
        }

        PixelData* pixelData = pixelDataPointers[row][col];
        if (pixelData) {
          if (pixelSet.find(*pixelData) == pixelSet.end()) {
            std::cout << "ERROR: Trying to erase an object from pixelSet that doesn't exist" << std::endl;
          }

          pixelSet.erase(*pixelData);
          pixelDataPointers[row][col] = nullptr;
        }
      }
    }
    extractedRegions.push_back(region);
  }

  return extractedRegions;
}

bool ExpansionRegionExtractor::expandRegion(std::vector<std::vector<int>>& fullRegion,
                                            Region& regionInfo,
                                            const int maxdist_x,
                                            const int maxdist_y,
                                            std::set<PixelData>& pixelSet,
                                            std::vector<std::vector<PixelData*>>& pixelDataPointers)
{
  int fullRegionHeight = fullRegion.size();
  int fullRegionWidth = fullRegion[0].size();

  int startRow = std::max(regionInfo.row - maxdist_x, 0);
  int startCol = std::max(regionInfo.col - maxdist_y, 0);
  int endRow = std::min(regionInfo.row + regionInfo.height + maxdist_x, fullRegionHeight);
  int endCol = std::min(regionInfo.col + regionInfo.width + maxdist_y, fullRegionWidth);

  bool regionExpanded = false;

  for (int row = startRow; row < endRow; ++row) {
    for (int col = startCol; col < endCol; ++col) {
      if (row >= regionInfo.row && row < regionInfo.row + regionInfo.height &&
          col >= regionInfo.col && col < regionInfo.col + regionInfo.width) {
        continue;
      }

      if (fullRegion[row][col] == 1) {
        if (col < regionInfo.col) {
          regionInfo.col = col;
          regionInfo.width += (regionInfo.col - col + 1);
        } else if (col >= regionInfo.col + regionInfo.width) {
          regionInfo.width += (col - regionInfo.col - regionInfo.width + 1);
        }

        if (row < regionInfo.row) {
          regionInfo.row = row;
          regionInfo.height += (regionInfo.row - row + 1);
        } else if (row >= regionInfo.row + regionInfo.height) {
          regionInfo.height += (row - regionInfo.row - regionInfo.height + 1);
        }
        regionExpanded = true;
      }
    }
  }
  return regionExpanded;
}

std::vector<std::vector<int>> ExpansionRegionExtractor::convertSparsePixelsToGrid(const std::vector<PixelData> pixels)
{
  uint16_t maxRow = 0, maxCol = 0;
  for (const auto& pixel : pixels) {
    maxRow = std::max(maxRow, pixel.getRow());
    maxCol = std::max(maxCol, pixel.getCol());
  }

  std::vector<std::vector<int>> grid(maxRow + 1, std::vector<int>(maxCol + 1, 0));

  for (const auto& pixel : pixels) {
    grid[pixel.getRow()][pixel.getCol()] = 1;
  }

  return grid;
}


#define CHECK_CUDA_ERROR(err)                                       \
  do {                                                              \
    if (err != hipSuccess) {                                       \
      fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err)); \
    }                                                               \
  } while (0)

namespace o2::itsmft::gpu
{
using o2::itsmft::Point;
using o2::itsmft::BoundingBox;

__device__ void print(int* data, int* labels, int* parent, int* rank, int nrow, int ncol)
{
  for (int i = 0; i < nrow; i++) {
    for (int j = 0; j < ncol; j++) {
      // Print elements of the Data Matrix
      printf("%d ", data[i * ncol + j]);
    }
    printf("\t\t");

    for (int j = 0; j < ncol; j++) {
      // Print elements of the Labels Matrix
      printf("%d ", labels[i * ncol + j]);
    }
    printf("\t\t");

    for (int j = 0; j < ncol; j++) {
      // Print elements of the Parent Matrix
      printf("%d ", parent[i * ncol + j]);
    }
    printf("\t\t");

    for (int j = 0; j < ncol; j++) {
      // Print elements of the Rank Matrix
      printf("%d ", rank[i * ncol + j]);
    }
    printf("\n");
  }
  printf("\n");
}

// recursively find root of cluster
__device__ int find(int x, int* parent)
{
  if (x != parent[x]) {
    parent[x] = find(parent[x], parent);
  }
  return parent[x];
}

// merge two clusters
__device__ int unify(int x, int y, int* parent, int* rank)
{
  int rootX = find(x, parent);
  int rootY = find(y, parent);

  if (rootX == rootY)
    return;

  if (rank[rootX] < rank[rootY]) {
    parent[rootX] = rootY;
    return rootY;
  } else {
    parent[rootY] = rootX;
    if (rank[rootX] == rank[rootY]) {
      rank[rootX]++;
    }
    return rootX;
  }
}

__global__ void ccl_kernel(int N, int* data, int* regionSizes, int* regionHeights, int* regionWidths,
                           int* startIndices, int* labels, int* parent, int* rank, int* numClusters,
                           int* clusterSizes, Point* scratchMemory, int* scratchMemIndex, int* stridedSizes, 
                           int* stridedPositions, int stride, BoundingBox* boxes, BoundingBox* stridedBoxes, 
                           int* chipIds, int* stridedChipIds)
{
  int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadIndex >= N) {
    return;
  }

  for (int regionIdx = threadIndex; regionIdx < N; regionIdx += blockDim.x * gridDim.x) {
    int startIndex = startIndices[regionIdx];
    int width = regionWidths[regionIdx];
    int height = regionHeights[regionIdx];

    int currentLabel = 1;

    // First pass
    for (int r = 0; r < height; ++r) {
      for (int c = 0; c < width; ++c) {
        int idx = startIndex + r * width + c;
        if (data[idx] == 1) {
          int leftIdx = (c == 0 ? -1 : idx - 1);
          int topIdx = (r == 0 ? -1 : idx - width);

          bool connectLeft = leftIdx != -1 && data[leftIdx] == 1;
          bool connectTop = topIdx != -1 && data[topIdx] == 1;

          if (!connectLeft && !connectTop) {
            labels[idx] = currentLabel++;
            parent[idx] = idx;
            rank[idx] = 0;
            numClusters[regionIdx]++;
            clusterSizes[idx] = 1;
            boxes[idx].min_r = r;
            boxes[idx].min_c = c;
            boxes[idx].max_r = r;
            boxes[idx].max_c = c;
          } else if (connectLeft && connectTop) {
            int topRootIdx = find(topIdx, parent);
            int leftRootIdx = find(leftIdx, parent);

            if (topRootIdx == leftRootIdx) {
              labels[idx] = labels[topRootIdx];
              parent[idx] = topRootIdx;
              rank[idx] = 0;

              // in this case, the bounding box already encompasses the newly added pixel
            } else {
              int newRoot = unify(leftRootIdx, topRootIdx, parent, rank);
              labels[idx] = labels[newRoot];
              parent[idx] = newRoot;
              numClusters[regionIdx]--;
              clusterSizes[newRoot] = clusterSizes[leftRootIdx] + clusterSizes[topRootIdx];

              boxes[newRoot].min_r = min(boxes[topRootIdx].min_r, boxes[leftRootIdx].min_r);
              boxes[newRoot].min_c = min(boxes[topRootIdx].min_c, boxes[leftRootIdx].min_c);
              boxes[newRoot].max_r = max(boxes[topRootIdx].max_r, boxes[leftRootIdx].max_r);
              boxes[newRoot].max_c = max(boxes[topRootIdx].max_c, boxes[leftRootIdx].max_c);
            }
          } else {
            int rootIdx;
            if (connectLeft) {
              rootIdx = find(leftIdx, parent);
            } else if (connectTop) {
              rootIdx = find(topIdx, parent);
            }
            labels[idx] = labels[rootIdx];
            parent[idx] = rootIdx;
            rank[idx] = 0;
            clusterSizes[rootIdx]++;

            boxes[rootIdx].min_r = min(boxes[rootIdx].min_r, r);
            boxes[rootIdx].min_c = min(boxes[rootIdx].min_c, c);
            boxes[rootIdx].max_r = max(boxes[rootIdx].max_r, r);
            boxes[rootIdx].max_c = max(boxes[rootIdx].max_c, c);
          }
        }
      }
    }

    int localClusterIndex = 0;

    // Second pass
    for (int r = 0; r < height; ++r) {
      for (int c = 0; c < width; ++c) {
        int idx = startIndex + r * width + c;
        if (data[idx] != 0) {
          int rootIdx = find(idx, parent);
          labels[idx] = labels[rootIdx];

          if (idx == rootIdx) {
            int currentClusterSize = clusterSizes[idx];
            BoundingBox currentClusterBox = boxes[idx];

            // global scratch memory index
            int reservedIndex = atomicAdd(scratchMemIndex, currentClusterSize);

            // pseudo local strided index
            int stridedIndex = stride * threadIndex + localClusterIndex;
            stridedSizes[stridedIndex] = currentClusterSize;
            stridedBoxes[stridedIndex] = currentClusterBox;
            stridedPositions[stridedIndex] = reservedIndex;
            stridedChipIds[stridedIndex] = chipIds[regionIdx];

            int localPixelIndex = 0;
            for (int r2 = 0; r2 < height; ++r2) {
              for (int c2 = 0; c2 < width; ++c2) {
                int idx2 = startIndex + r2 * width + c2;
                if (data[idx2] != 0 && find(idx2, parent) == rootIdx) {
                  scratchMemory[reservedIndex + localPixelIndex].r = r2;
                  scratchMemory[reservedIndex + localPixelIndex].c = c2;
                  localPixelIndex++;
                }
              }
            }
          }
          localClusterIndex++;
        }
      }
    }
  }
}
} // namespace o2::itsmft::gpu

std::vector<int> flatten(const std::vector<std::vector<std::vector<int>>>& data)
{
  std::vector<int> flatData;
  for (const auto& region : data)
    for (const auto& row : region)
      for (const auto& pixel : row)
        flatData.push_back(pixel);
  return flatData;
}

void PlayneClusterAlgorithm::clusterize(const std::vector<std::vector<std::vector<int>>>& data, const std::vector<int>& chipIds, std::vector<Clusterer::BBox>& clusterBBoxes, std::vector<std::vector<PixelData>>& clusterPixels)
{
  std::vector<int> flatData = flatten(data);

  int totalNumPixels = flatData.size();
  int numRegions = data.size();
  int scratchMemLength = 100 * numRegions; // assuming that there will not be more than 100 pixels in clusters per region on average
  int stride = 8;

  std::vector<int> regionSizes;
  std::vector<int> regionHeights;
  std::vector<int> regionWidths;
  std::vector<int> startIndices;
  int* labels = new int[totalNumPixels]();
  int* parent = new int[totalNumPixels]();
  int* rank = new int[totalNumPixels]();
  int* clusterSizes = new int[totalNumPixels]();
  int* numClusters = new int[numRegions]();

  Point* scratchMemory = new Point[scratchMemLength]();
  int* stridedSizes = new int[numRegions * stride]();
  int* stridedPositions = new int[numRegions * stride]();
  int scratchMemIndex = 0;
  BoundingBox* boxes = new BoundingBox[totalNumPixels]();
  BoundingBox* stridedBoxes = new BoundingBox[numRegions * stride]();
  int* stridedChipIds = new int[numRegions * stride]();

  // change that stuff to hipMemset
  for (int i = 0; i < totalNumPixels; i++) {
    parent[i] = i;
    labels[i] = 0;
    rank[i] = 0;
    clusterSizes[i] = 0;

    boxes[i].min_r = 0;
    boxes[i].max_r = 0;
    boxes[i].min_c = 0;
    boxes[i].max_c = 0;
  }

  for (int i = 0; i < numRegions; i++) {
    numClusters[i] = 0;
  }

  for (int i = 0; i < numRegions * stride; i++) {
    stridedSizes[i] = 0;
    stridedPositions[i] = 0;
    stridedChipIds[i] = 0;
  }

  for (int i = 0; i < scratchMemLength; i++) {
    scratchMemory[i].r = -1;
    scratchMemory[i].c = -1;
  }

  for (int i = 0; i < numRegions * stride; i++) {
    stridedBoxes[i].min_r = 0;
    stridedBoxes[i].max_r = 0;
    stridedBoxes[i].min_c = 0;
    stridedBoxes[i].max_c = 0;
  }

  startIndices.push_back(0);

  for (const auto& region : data) {
    regionSizes.push_back(region.size() * (region.empty() ? 0 : region[0].size()));
    regionHeights.push_back(region.size());
    regionWidths.push_back((region.empty() ? 0 : region[0].size()));
    startIndices.push_back(startIndices.back() + regionSizes.back());
  }

  // Remove the last start index (beyond the end of the data)
  startIndices.pop_back();

  int* deviceData;
  int* deviceChipIds;
  int* deviceSizes;
  int* deviceHeights;
  int* deviceWidths;
  int* deviceStartIndices;
  int* deviceLabels;
  int* deviceParents;
  int* deviceRanks;
  int* deviceClusterSizes;
  int* deviceNumClusters;

  Point* deviceScratchMemory;
  int* deviceScratchMemIndex;
  int* deviceStridedSizes;
  int* deviceStridedPositions;
  int* deviceStridedChipIds;
  BoundingBox* deviceBoxes;
  BoundingBox* deviceStridedBoxes;

  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceData, totalNumPixels * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceChipIds, numRegions * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceSizes, regionSizes.size() * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceHeights, regionHeights.size() * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceWidths, regionWidths.size() * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceStartIndices, startIndices.size() * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceLabels, totalNumPixels * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceParents, totalNumPixels * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceRanks, totalNumPixels * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceClusterSizes, totalNumPixels * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceNumClusters, numRegions * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceScratchMemory, scratchMemLength * sizeof(Point)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceScratchMemIndex, sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceStridedSizes, stride * numRegions * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceStridedPositions, stride * numRegions * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceStridedChipIds, stride * numRegions * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceBoxes, totalNumPixels * sizeof(BoundingBox)));
  CHECK_CUDA_ERROR(hipMalloc((void**)&deviceStridedBoxes, stride * numRegions * sizeof(BoundingBox)));

  CHECK_CUDA_ERROR(hipMemcpy(deviceData, flatData.data(), totalNumPixels * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceChipIds, chipIds.data(), numRegions * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceSizes, regionSizes.data(), regionSizes.size() * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceWidths, regionWidths.data(), regionWidths.size() * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceHeights, regionHeights.data(), regionHeights.size() * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceStartIndices, startIndices.data(), startIndices.size() * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceLabels, labels, totalNumPixels * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceParents, parent, totalNumPixels * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceRanks, rank, totalNumPixels * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceClusterSizes, clusterSizes, totalNumPixels * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceNumClusters, numClusters, numRegions * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceScratchMemory, scratchMemory, scratchMemLength * sizeof(Point), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceScratchMemIndex, &scratchMemIndex, sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceStridedSizes, stridedSizes, stride * numRegions * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceStridedPositions, stridedPositions, stride * numRegions * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceStridedChipIds, stridedChipIds, stride * numRegions * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceBoxes, boxes, totalNumPixels * sizeof(BoundingBox), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(deviceStridedBoxes, stridedBoxes, stride * numRegions * sizeof(BoundingBox), hipMemcpyHostToDevice));

  const int threadsPerBlock = 256;
  const int numBlocks = (numRegions + threadsPerBlock - 1) / threadsPerBlock;

  gpu::ccl_kernel<<<numBlocks, threadsPerBlock>>>(numRegions, deviceData, deviceSizes, deviceHeights,
                                                  deviceWidths, deviceStartIndices, deviceLabels, deviceParents, deviceRanks,
                                                  deviceNumClusters, deviceClusterSizes, deviceScratchMemory, deviceScratchMemIndex,
                                                  deviceStridedSizes, deviceStridedPositions, stride, deviceBoxes, deviceStridedBoxes, 
                                                  deviceChipIds, deviceStridedChipIds);

  CHECK_CUDA_ERROR(hipMemcpy(labels, deviceLabels, totalNumPixels * sizeof(int), hipMemcpyDeviceToHost));
  CHECK_CUDA_ERROR(hipMemcpy(scratchMemory, deviceScratchMemory, scratchMemLength * sizeof(int), hipMemcpyDeviceToHost)); // copying can be shortened for tiny optimization
  CHECK_CUDA_ERROR(hipMemcpy(stridedSizes, deviceStridedSizes, stride * numRegions * sizeof(int), hipMemcpyDeviceToHost));
  CHECK_CUDA_ERROR(hipMemcpy(stridedPositions, deviceStridedPositions, stride * numRegions * sizeof(int), hipMemcpyDeviceToHost));
  CHECK_CUDA_ERROR(hipMemcpy(stridedChipIds, deviceStridedChipIds, stride * numRegions * sizeof(int), hipMemcpyDeviceToHost));
  CHECK_CUDA_ERROR(hipMemcpy(stridedBoxes, deviceStridedBoxes, stride * numRegions * sizeof(BoundingBox), hipMemcpyDeviceToHost));

  for (int clusterIdx = 0; clusterIdx < stride * numRegions; ++clusterIdx) {
    if (stridedSizes[clusterIdx] == 0)
      continue;

    clusterPixels.emplace_back();
    int clusterStartPosition = stridedPositions[clusterIdx];
    int chipId = stridedChipIds[clusterIdx];

    const BoundingBox& sBox = stridedBoxes[clusterIdx];
    Clusterer::BBox bBox(chipId);
    bBox.rowMin = static_cast<uint16_t>(sBox.min_r);
    bBox.colMin = static_cast<uint16_t>(sBox.min_c);
    bBox.rowMax = static_cast<uint16_t>(sBox.max_r);
    bBox.colMax = static_cast<uint16_t>(sBox.max_c);
    clusterBBoxes.push_back(bBox);

    for (int pixelIdx = 0; pixelIdx < stridedSizes[clusterIdx]; ++pixelIdx) {
      clusterPixels.back().push_back(PixelData(scratchMemory[clusterStartPosition + pixelIdx].r, scratchMemory[clusterStartPosition + pixelIdx].c));
    }
  }

  for (int i = 0; i < std::min(static_cast<int>(clusterPixels.size()), 20); i++) {
    for (const PixelData& pixel : clusterPixels[i]) {
      std::cout << "(" << pixel.getRow() << "," << pixel.getCol() << ") ";
    }
    std::cout << std::endl;
  }

  for (int i = 0; i < std::min(static_cast<int>(clusterBBoxes.size()), 20); i++) {
    std::cout << "(" << clusterBBoxes[i].chipID << "," << clusterBBoxes[i].rowMin << "," << clusterBBoxes[i].rowMax << "," << clusterBBoxes[i].colMin << "," << clusterBBoxes[i].colMax << ") "; 
    std::cout << std::endl;
  }

  CHECK_CUDA_ERROR(hipFree(deviceData));
  CHECK_CUDA_ERROR(hipFree(deviceSizes));
  CHECK_CUDA_ERROR(hipFree(deviceWidths));
  CHECK_CUDA_ERROR(hipFree(deviceHeights));
  CHECK_CUDA_ERROR(hipFree(deviceStartIndices));
  CHECK_CUDA_ERROR(hipFree(deviceLabels));
  CHECK_CUDA_ERROR(hipFree(deviceParents));
  CHECK_CUDA_ERROR(hipFree(deviceRanks));

  delete[] labels;
  delete[] parent;
  delete[] rank;
}